#include "error.cuh"
#include "memory.h"
#include <stdlib.h>

void allocate_memory(int N, int MN, Atom *atom)
{
    CHECK(hipMallocManaged((void**)&atom->NN, sizeof(int) * N))
    CHECK(hipMallocManaged((void**)&atom->NL, sizeof(int) * N * MN))
    CHECK(hipMallocManaged((void**)&atom->m, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->x, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->y, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->z, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->vx, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->vy, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->vz, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->fx, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->fy, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->fz, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->pe, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->ke, sizeof(real) * N))
    CHECK(hipMallocManaged((void**)&atom->box, sizeof(real) * 6))
}

void deallocate_memory(Atom *atom)
{
    CHECK(hipFree(atom->NN)) 
    CHECK(hipFree(atom->NL))
    CHECK(hipFree(atom->m))
    CHECK(hipFree(atom->x))
    CHECK(hipFree(atom->y))
    CHECK(hipFree(atom->z))
    CHECK(hipFree(atom->vx))
    CHECK(hipFree(atom->vy))
    CHECK(hipFree(atom->vz))
    CHECK(hipFree(atom->fx))
    CHECK(hipFree(atom->fy))
    CHECK(hipFree(atom->fz))
    CHECK(hipFree(atom->pe))
    CHECK(hipFree(atom->ke))
    CHECK(hipFree(atom->box))
}

