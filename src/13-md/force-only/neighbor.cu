#include "hip/hip_runtime.h"
#include "neighbor.h"
#include "mic.h"
#include <stdio.h>
#include <stdlib.h>

static void __global__ gpu_find_neighbor
(
    int N, int *g_NN, int *g_NL, real *g_box, 
    real *g_x, real *g_y, real *g_z, real cutoff2
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {
        int count = 0;
        real x1 = g_x[n1];
        real y1 = g_y[n1];
        real z1 = g_z[n1];
        for (int n2 = 0; n2 < N; n2++)
        {
            real x12 = g_x[n2] - x1;
            real y12 = g_y[n2] - y1;
            real z12 = g_z[n2] - z1;
            apply_mic(g_box, &x12, &y12, &z12);
            real d12_square = x12*x12 + y12*y12 + z12*z12;
            if ((n2 != n1) && (d12_square < cutoff2))
            {
                g_NL[count++ * N + n1] = n2;
            }
        }
        g_NN[n1] = count;
    }
}

void find_neighbor(int N, int MN, Atom *atom)
{
    real cutoff = 11.0;
    real cutoff2 = cutoff * cutoff;

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    gpu_find_neighbor<<<grid_size, block_size>>>
    (
        N, atom->g_NN, atom->g_NL, atom->g_box,
        atom->g_x, atom->g_y, atom->g_z, cutoff2
    );
}

